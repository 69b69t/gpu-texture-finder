#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#define IS_112 0
#define UNKNOWN_ROTATION 1

struct Pos3d{
    int32_t x;
    int32_t y;
    int32_t z;
    uint8_t rotation;
};

__device__ static inline int32_t random(long seed) {
    seed = (seed ^ 0x5DEECE66DULL) & ((1ULL << 48) - 1);
    return (int)((seed * 0xBB20B4600A69ULL + 0x40942DE6BAULL) >> 16);
}

__device__ static inline int32_t getRotation(const int32_t x, const int32_t y, const int32_t z)
{
    /*
    gets the rotation at a block
    */
    int64_t i = (int64_t)(int32_t)(3129871ULL * (uint32_t)x) ^ (int64_t)((uint64_t)z * 116129781ULL) ^ (int64_t)y;
    i = i * i * 42317861ULL + i * 11ULL;
    
    //int cast for 1.12-, otherwise none
    if(!IS_112) i = i >> 16;
    else i = (int)i >> 16;

    //no random call in 1.12-
    if(!IS_112) return abs(random(i)) % 4;
    else return abs(i) % 4;
}

__device__ static inline void rotate90DegCW(struct Pos3d* formation, uint32_t formationCount)
{
    /*
    rotates a formation 90 degrees clockwise
    */
    int32_t temp;
    for(uint32_t i = 0; i < formationCount; i++)
    {
        //(x,z) rotated 90 deg would be (z,-x)
        //swap x and z then negate z
        temp = formation[i].z;
        formation[i].z = -formation[i].x;
        formation[i].x = temp;
        //printf("formation[i].z = %d\n", formation[i].z);
    }
}

__device__ static inline uint32_t checkFormation(struct Pos3d* formation, uint32_t formationCount, int32_t x, int32_t y, int32_t z)
{
    /*
    takes an x,y,z position and a formation and checks ONE orientation
    returns 0 if not a match, and 1 if there is
    */
    for(uint32_t i = 0; i < formationCount; i++)
    {
        //if block rotation is not equal to the rotation we're searching for, invalid
        if(getRotation(x+formation[i].x, y+formation[i].y, z+formation[i].z) != ((formation[i].rotation) % 4)) return 0;
    }
    return 1;
}

/*
best times
4 rots: 33.0
1 rot: 10.1
*/

__device__ static inline int32_t isMatching(int32_t x, int32_t y, int32_t z)
{
    /*
    takes in a x,y,z position and returns 0 if it dosent match or 1 if it does
    */


    //change these and reference the definition of Pos3d
    uint32_t formationCount = 4;
    struct Pos3d offsets[] = {
        {0, 0, 0, 3}, //rotation 3 at reference point (x,y,z)
        {1, 0, 0, 0}, //rotation 0 at point relative (x+1,y,z)
        {0, 2, 0, 0}, //rotation 0 at point (x,y+2,z)
        {3, 3, 2, 1} //rotation 1 at point (x+3,y+3,z+2)
    };

    uint32_t loops;
    if(UNKNOWN_ROTATION) loops = 4;
    else loops = 1;

    for(uint32_t j = 0; j < loops; j++)
    {
        //if checkFormation ever returns 1, we have a match and can print it
        if(checkFormation(formation, formationCount, x, y, z)) return 1;

        //rotate
        rotate90DegCW(formation, formationCount);
    }

    //else is invalid
    return 0;
}

__global__ void spawnThread(const int32_t xMin, const int32_t xMax, const int32_t zMin, const int32_t zMax, const int32_t yPos)
{
    /*
    spawns a single searcher thread that is aware of which thread it is
    */
    const int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t xRange = xMax - xMin;
    const int32_t zRange = zMax - zMin;

    uint32_t xPos;
    uint32_t zPos;

    //"< (xRange*zRange)"" because the bottomright most item is just the amount of positions to check
    for(int64_t position = threadId; position < ((uint64_t)xRange*(uint64_t)zRange); position += ( blockDim.x * gridDim.x ))
    {
        xPos = position%xRange + xMin;
        zPos = position/xRange + zMin;

        if(isMatching(xPos, yPos, zPos)) printf("%d,%d,%d\n", xPos, yPos, zPos);
    }

}

int main()
{
    hipError_t err;

    int32_t xMin = -1000000;
    int32_t zMin = -100000;

    int32_t xMax = 100000;
    int32_t zMax = 100000;

    int32_t yMin = 63;
    int32_t yMax = 63;

    for(; yMin <= yMax; yMin++)
    {
        //if you get
        //Error: too many resources requested for launch
        //decrease these parameters (the 1024s)
        spawnThread<<<1024,1024>>>(xMin, xMax, zMin, zMax, yMin);

        //error checking
        err = hipGetLastError();
        if(err != hipSuccess){printf("Error: %s\n", hipGetErrorString(err));exit(-1);}
        hipDeviceSynchronize();
        printf("complete with y=%d\n", yMin);
    }
    printf("complete\n");
}
