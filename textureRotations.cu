
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

__device__ static inline int32_t random(long seed) {
    seed = (seed ^ 0x5DEECE66DL) & ((1L << 48) - 1);
    return (int)((seed * 0xBB20B4600A69L + 0x40942DE6BAL) >> 16);
}

__device__ static inline int32_t getRotation(const int32_t x, const int32_t y, const int32_t z)
{
    int64_t i = (int64_t)(int32_t)(3129871U * (uint32_t)x) ^ (int64_t)((uint64_t)z * 116129781ULL) ^ (int64_t)y;
    i = i * i * 42317861ULL + i * 11ULL;
    i = i >> 16;
    return abs(random(i)) % 4;
}

__device__ static inline int32_t isMatching(int32_t x, int32_t y, int32_t z)
{
    //hardcode rotation values. right now its set to find squares of 5x5 blocks
    //with the same texture rotations. (actually its just a single direction but too lazy to change)

    for(uint32_t j = 0; j < 5; j++)
    {
        for(uint32_t i = 0; i < 5; i++)
        {
            //set y value here
            if(getRotation(x+i, y, z+j) != 0) return 0;
        }
    }
    return 1;
}

//get id, that will be offset
//take a number and use it as counter
//each thread goes over a list of a bunch of positions
//where (x = counter/xRange)
//and (z = counter % zRange)
//once a thread reaches a position outside of the bounds, kill the thread
//thread will always go out of bounds on xMax

__global__ void spawnThread(const int32_t xMin, const int32_t xMax, const int32_t zMin, const int32_t zMax, const int32_t yPos)
{
    //each gpu thread will search a single band, from (x, -30000000) to (x, 30000000)
    //60 mil checks each
    const int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t xRange = xMax - xMin;
    const int32_t zRange = zMax - zMin;

    uint32_t xPos;
    uint32_t zPos;

    //< (xRange*zRange) because the bottomright most item is just the amount of positions to check
    for(int64_t position = threadId; position < ((uint64_t)xRange*(uint64_t)zRange); position += (1024*1024))
    {
        xPos = position%xRange + xMin;
        zPos = position/xRange + zMin;

        //without using the +xMin +zMin, the code always puts the searchy thing at 
        if(isMatching(xPos, yPos, zPos)) printf("%d,%d,%d\n", xPos, yPos, zPos);
        //if(zPos==0)printf("%d,%d,%d checked pos:%ld\n", xPos, yPos, zPos, position);

    }

}

int main()
{
    hipError_t err;

    int32_t xMin = -50000;
    int32_t zMin = -50000;

    int32_t xMax = 50000;
    int32_t zMax = 50000;

    int32_t yMin = 0;
    int32_t yMax = 256;

    //spawnThread<<<1024,1024>>>(xMin, xMax, zMin, zMax, 64);
    //cudaDeviceSynchronize();
    //return 0;

    for(; yMin < yMax; yMin++)
    {
        spawnThread<<<1024,1024>>>(xMin, xMax, zMin, zMax, yMin);

        //error checking
        err = hipGetLastError();
        if(err != hipSuccess){printf("Error: %s\n", hipGetErrorString(err));exit(-1);}
        printf("complete with y=%d\n", yMin);
        hipDeviceSynchronize();
    }
    printf("complete\n");
}
